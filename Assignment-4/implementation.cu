/*
============================================================================
Filename    : algorithm.c
Author      : Olivier Cloux
SCIPER      : 236079
============================================================================
*/

#include <iostream>
#include <iomanip>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
using namespace std;

// CPU Baseline
void array_process(double *input, double *output, int length, int iterations)
{
  double *temp;
  for(int n=0; n<(int) iterations; n++)
  {
    for(int i=1; i<length-1; i++)
    {
      for(int j=1; j<length-1; j++)
      {
        output[(i)*(length)+(j)] = (input[(i-1)*(length)+(j-1)] +
        input[(i-1)*(length)+(j)]   +
        input[(i-1)*(length)+(j+1)] +
        input[(i)*(length)+(j-1)]   +
        input[(i)*(length)+(j)]     +
        input[(i)*(length)+(j+1)]   +
        input[(i+1)*(length)+(j-1)] +
        input[(i+1)*(length)+(j)]   +
        input[(i+1)*(length)+(j+1)] ) / 9;

      }
    }
    output[(length/2-1)*length+(length/2-1)] = 1000;
    output[(length/2)*length+(length/2-1)]   = 1000;
    output[(length/2-1)*length+(length/2)]   = 1000;
    output[(length/2)*length+(length/2)]     = 1000;

    temp = input;
    input = output;
    output = temp;
  }
}

__global__ void heattransfer(double *input, double *output, int length)
{
  int i = blockIdx.x + 1;
  int j = threadIdx.x + 1;
  output[i*length + j] = (input[(i-1)*(length)+(j-1)] +
                      input[(i-1)*(length)+(j)]   +
                      input[(i-1)*(length)+(j+1)] +
                      input[(i)*(length)+(j-1)]   +
                      input[(i)*(length)+(j)]     +
                      input[(i)*(length)+(j+1)]   +
                      input[(i+1)*(length)+(j-1)] +
                      input[(i+1)*(length)+(j)]   +
                      input[(i+1)*(length)+(j+1)] ) / 9;

  output[(length/2-1)*length+(length/2-1)] = 1000;
  output[(length/2)*length+(length/2-1)]   = 1000;
  output[(length/2-1)*length+(length/2)]   = 1000;
  output[(length/2)*length+(length/2)]     = 1000;
}

// GPU Optimized function
void GPU_array_process(double *input, double *output, int length, int iterations)
{
  //Cuda events for calculating elapsed time
  hipEvent_t cpy_H2D_start, cpy_H2D_end, comp_start, comp_end, cpy_D2H_start, cpy_D2H_end;
  hipEventCreate(&cpy_H2D_start);
  hipEventCreate(&cpy_H2D_end);
  hipEventCreate(&cpy_D2H_start);
  hipEventCreate(&cpy_D2H_end);
  hipEventCreate(&comp_start);
  hipEventCreate(&comp_end);

  /* Preprocessing goes here */
  size_t size = length*length*sizeof(double);
  double* in;
  double* out;
  double* temp;
  if(hipMalloc((void **) &in, size) != hipSuccess){
    return;
  }
  if(hipMalloc((void **) &out, size) != hipSuccess){
    hipFree(in);
    return;
  }

  hipEventRecord(cpy_H2D_start);
  /* Copying array from host to device goes here */
  hipMemcpy(in, input, size, hipMemcpyHostToDevice);

  hipEventRecord(cpy_H2D_end);
  hipEventSynchronize(cpy_H2D_end);

  hipEventRecord(comp_start);
  /*GPU computation*/
  for (size_t k = 0; k < iterations; k++) {
    /*Starting a kernel with each iteration to ensure synchronization
    * between blocks*/
    heattransfer<<<length-2, length-2>>>(in, out, length);

    temp = in;
    in = out;
    out = temp;
  }
  hipEventRecord(comp_end);
  hipEventSynchronize(comp_end);

  hipEventRecord(cpy_D2H_start);
  /*copy result from device to host*/
  if(iterations%2 == 0){
    hipMemcpy(output, in, size, hipMemcpyDeviceToHost);
  } else{
    hipMemcpy(output, out, size, hipMemcpyDeviceToHost);
  }

  hipEventRecord(cpy_D2H_end);
  hipEventSynchronize(cpy_D2H_end);

  hipFree(in);
  hipFree(out);

  float time;
  hipEventElapsedTime(&time, cpy_H2D_start, cpy_H2D_end);
  cout<<"Host to Device MemCpy takes "<<setprecision(4)<<time/1000<<"s"<<endl;

  hipEventElapsedTime(&time, comp_start, comp_end);
  cout<<"Computation takes "<<setprecision(4)<<time/1000<<"s"<<endl;

  hipEventElapsedTime(&time, cpy_D2H_start, cpy_D2H_end);
  cout<<"Device to Host MemCpy takes "<<setprecision(4)<<time/1000<<"s"<<endl;
}
